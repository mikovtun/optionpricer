#include "hip/hip_runtime.h"
#include <iostream>
#include <random>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stock.h>
#include <device_properties.h>

namespace OP {


// Generates log(a + b*norm_rand)
__global__ void gen_prices(double* nums, size_t n, const double start, const double logMean, const double logStdDev, unsigned long seed) {
  size_t id = threadIdx.x + blockIdx.x * blockDim.x;

  if( id < n ) {
    hiprandState state;
    hiprand_init(seed, id, 0, &state);
    nums[id] = start * hiprand_log_normal_double(&state, logMean, logStdDev);
  }
}

__global__ void sumReduce(double* input, float* output) {
  size_t id = 2*threadIdx.x;
  for( size_t stride = 1; stride <= blockDim.x; stride *= 2) {
    if( threadIdx.x % stride == 0 )
      input[id] += input[id + stride];
    __syncthreads();
  }
  if(threadIdx.x == 0)
    *output = input[0];
}



template <>
double LogNormalStock<Device::gpu>::getPrice(float u) {
  std::cout << "test" << std::endl;
  return 0.0;
}

template <>
void LogNormalStock<Device::gpu>::getPrices(size_t N, double* out, float u) {
  std::random_device rd;

  //const double vol2 = volatility*volatility;
  //const double logMean = (bias - vol2*0.5)*u;
  //const double logStdDev = vol2*u;
  const double vol2 = volatility*volatility;
  const double logMean = bias * u - 0.5*vol2*u;
  const double logStdDev = volatility*sqrt(u);

  auto gpuInstance = GPUProperties::get();
  auto gpuProps = gpuInstance->getProperties();

  size_t maxThreadsPerBlock = gpuProps.maxThreadsPerBlock;
  size_t blocksPerRun = (N + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
  size_t freeMem = gpuInstance->getFreeMem();

  double* p_dev;
  hipError_t error = hipMalloc( &p_dev, N*sizeof(double));

  if( error != hipSuccess) 
    std::cout << hipGetErrorString(error) << std::endl;
  
  gen_prices<<<blocksPerRun, maxThreadsPerBlock>>>( p_dev, N, start, logMean, logStdDev, rd());

  error = hipMemcpy( out, p_dev, N*sizeof(double), hipMemcpyDeviceToHost);

  if( error != hipSuccess) 
    std::cout << hipGetErrorString(error) << std::endl;

  error = hipFree( p_dev);

  if( error != hipSuccess) 
    std::cout << hipGetErrorString(error) << std::endl;
  
}


}
