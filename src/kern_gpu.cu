#include "hip/hip_runtime.h"
#include <iostream>
#include <random>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stock.h>
#include <device_properties.h>

namespace OP {


// Generates log(a + b*norm_rand)
__global__ void gen_prices(float* nums, size_t n, const float start, const float logMean, const float logStdDev, unsigned long seed) {
  size_t id = threadIdx.x + blockIdx.x * blockDim.x;

  if( id < n ) {
    hiprandState state;
    hiprand_init(seed, id, 0, &state);
    nums[id] = start * hiprand_log_normal(&state, logMean, logStdDev);
  }
}

__global__ void sumReduce(float* input, float* output) {
  size_t id = 2*threadIdx.x;
  for( size_t stride = 1; stride <= blockDim.x; stride *= 2) {
    if( threadIdx.x % stride == 0 )
      input[id] += input[id + stride];
    __syncthreads();
  }
  if(threadIdx.x == 0)
    *output = input[0];
}


template <>
void LN<Device::gpu>::getLNModelPrices(size_t N, float* out, float u, float start, float bias, float volatility) {
  std::random_device rd;

  //const float vol2 = volatility*volatility;
  //const float logMean = (bias - vol2*0.5)*u;
  //const float logStdDev = vol2*u;
  const float vol2 = volatility*volatility;
  const float logMean = bias * u - 0.5*vol2*u;
  const float logStdDev = volatility*sqrt(u);

  auto gpuInstance = GPUProperties::get();
  auto gpuProps = gpuInstance->getProperties();

  size_t freeMem = gpuInstance->getFreeMem();
  //std::cout << "freeMem = " << freeMem << std::endl;
  //std::cout << "Can fit " << freeMem / sizeof(float) << " nums in memory" << std::endl;
  //N = (size_t)(freeMem / sizeof(float) * 0.95);

  size_t maxThreadsPerBlock = gpuProps.maxThreadsPerBlock;
  size_t blocksPerRun = (N + maxThreadsPerBlock - 1) / maxThreadsPerBlock;


  float* p_dev;
  hipError_t error = hipMalloc( &p_dev, N*sizeof(float));

  if( error != hipSuccess) 
    std::cout << hipGetErrorString(error) << std::endl;
  
  gen_prices<<<blocksPerRun, maxThreadsPerBlock>>>( p_dev, N, start, logMean, logStdDev, rd());

  error = hipMemcpy( out, p_dev, N*sizeof(float), hipMemcpyDeviceToHost);

  if( error != hipSuccess) 
    std::cout << hipGetErrorString(error) << std::endl;

  error = hipFree( p_dev);

  if( error != hipSuccess) 
    std::cout << hipGetErrorString(error) << std::endl;
  
}


}
